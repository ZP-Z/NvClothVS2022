#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2020 NVIDIA Corporation. All rights reserved.
// Copyright (c) 2004-2008 AGEIA Technologies, Inc. All rights reserved.
// Copyright (c) 2001-2004 NovodeX AG. All rights reserved.

#include "CuSolverKernel.h"
#include "CuClothData.h"
#include "CuPhaseConfig.h"

#include <new> // placement new

/*
	For detailed comments about the algorithm check SwSolverKernel.cpp (or the documentation)
	The CPU implementation is generally easier to read, and comments are not duplicated in other implementations.
	Only CUDA implementation specific comments are left in this implementation.
*/

#ifndef FLT_EPSILON
#define FLT_EPSILON 1.192092896e-07F
#endif
#ifndef FLT_MAX
#define FLT_MAX 3.402823466e+38F
#endif

// Converting pointers to shared/global addresses is faster than doing generic loads on SM50
#define CONVERT_ADDRESSES (__CUDA_ARCH__ >= 500)

#if !defined(_WIN64) && !defined(__x86_64__)
#define POINTER_CONSTRAINT "r"
#define POINTER_TYPE "u32"
#else
#define POINTER_CONSTRAINT "l"
#define POINTER_TYPE "u64"
#endif

#ifndef __CUDA_ARCH__
#define assert(x)
#endif

extern "C" {
extern _CRTIMP __host__ __device__ int __cdecl printf(const char*, ...);
}

using namespace nv;

// shared memory copy (instead of relying on constant cache)
__shared__ cloth::CuClothData gClothData;
__shared__ cloth::CuFrameData gFrameData;
__shared__ cloth::CuIterationData gIterData;

// Our way to create stream local variables
__shared__ void* gProfileBuffer;
__shared__ uint32_t gProfileBaseId;

static const uint32_t gCuClothDataSize = sizeof(cloth::CuClothData) / sizeof(float);
static const uint32_t gCuFrameDataSize = sizeof(cloth::CuFrameData) / sizeof(float);
static const uint32_t gCuIterationDataSize = sizeof(cloth::CuIterationData) / sizeof(float);
static const uint32_t gCuPhaseConfigSize = sizeof(cloth::CuPhaseConfig) / sizeof(float);

/*
Memory block for all temporary data in shared memory (in 'allocation' order).
The numbers indicate the allocation slot if used a stack allocator.
0) simulate*()::configs (numPhases*sizeof(CuPhaseConfig))
1) simulate*()::particles ({0,1,2}*4*numParticles floats)
2) CuCollision::mCapsuleIndices, mCapsuleMasks, mConvexMasks (numCapsules*4+numConvexes ints)
3) CuCollision::mPrevData (4*numSpheres+10*numCones floats)
4) CuCollision::collideConvexes() (4*numPlanes floats)
4) CuCollision::collideTriangles() (19*numTriangles floats)
4) CuCollision::mCurData::Spheres (4*numSpheres floats)
5) computeParticleBounds()::dst (192 floats written, 208 float read)
5) computeSphereBounds()::dst (192 floats written, 208 floats read)
5) CuCollision::mCurData::Cones (10*numCones floats)
6) CuCollision::mShapeGrid (2*6*sGridSize=96 floats)
4) CuSelfCollision::buildAcceleration()::buffer (34*16=544 ints)
*/
extern __shared__ float gSharedMemory[];
extern __shared__ int32_t gSharedSigned[];
extern __shared__ uint32_t gSharedUnsigned[];

/***************** Pointer Wrappers **********************/
enum AddressSpace
{
	Shared,
	Global
};

template <AddressSpace, typename T>
__device__ T load(const T* ptr);
template <AddressSpace, typename T>
__device__ void store(T* ptr, const T& value);

#if !CONVERT_ADDRESSES
template <AddressSpace, typename T>
__device__ T load(const T* ptr)
{
	return *ptr;
}
template <AddressSpace, typename T>
__device__ void store(T* ptr, const T& value)
{
	*ptr = value;
}
#else
template <>
__device__ float load<Shared>(const float* ptr)
{
	float value;
	asm("ld.shared.f32 %0, [%1];" : "=f"(value) : POINTER_CONSTRAINT(ptr));
	return value;
}
template <>
__device__ int32_t load<Shared>(const int32_t* ptr)
{
	int32_t value;
	asm("ld.shared.s32 %0, [%1];" : "=r"(value) : POINTER_CONSTRAINT(ptr));
	return value;
}
template <>
__device__ uint32_t load<Shared>(const uint32_t* ptr)
{
	uint32_t value;
	asm("ld.shared.u32 %0, [%1];" : "=r"(value) : POINTER_CONSTRAINT(ptr));
	return value;
}
template <>
__device__ void store<Shared>(int32_t* ptr, const int32_t& value)
{
	asm("st.shared.s32 [%0], %1;" : : POINTER_CONSTRAINT(ptr), "r"(value) : "memory");
}
template <>
__device__ void store<Shared>(float* ptr, const float& value)
{
	asm("st.shared.f32 [%0], %1;" : : POINTER_CONSTRAINT(ptr), "f"(value) : "memory");
}
template <>
__device__ void store<Shared>(uint32_t* ptr, const uint32_t& value)
{
	asm("st.shared.u32 [%0], %1;" : : POINTER_CONSTRAINT(ptr), "r"(value) : "memory");
}
template <>
__device__ float load<Global>(const float* ptr)
{
	float value;
	asm("ld.global.f32 %0, [%1];" : "=f"(value) : POINTER_CONSTRAINT(ptr));
	return value;
}
template <>
__device__ int32_t load<Global>(const int32_t* ptr)
{
	int32_t value;
	asm("ld.global.s32 %0, [%1];" : "=r"(value) : POINTER_CONSTRAINT(ptr));
	return value;
}
template <>
__device__ uint32_t load<Global>(const uint32_t* ptr)
{
	uint32_t value;
	asm("ld.global.u32 %0, [%1];" : "=r"(value) : POINTER_CONSTRAINT(ptr));
	return value;
}
template <>
__device__ void store<Global>(int32_t* ptr, const int32_t& value)
{
	asm("st.global.s32 [%0], %1;" : : POINTER_CONSTRAINT(ptr), "r"(value) : "memory");
}
template <>
__device__ void store<Global>(float* ptr, const float& value)
{
	asm("st.global.f32 [%0], %1;" : : POINTER_CONSTRAINT(ptr), "f"(value) : "memory");
}
template <>
__device__ void store<Global>(uint32_t* ptr, const uint32_t& value)
{
	asm("st.global.u32 [%0], %1;" : : POINTER_CONSTRAINT(ptr), "r"(value) : "memory");
}
#endif

template <AddressSpace, typename>
struct Pointer;

template <AddressSpace S, typename T>
struct Reference
{
	template <AddressSpace, typename>
	friend struct Reference;
	friend struct Pointer<S, T>;

	__device__ Reference()
	{
	}
	__device__ Reference(const Reference& other) : mPtr(other.mPtr)
	{
	}
	template <typename U>
	__device__ Reference(const Reference<S, U>& other)
	: mPtr(other.mPtr)
	{
	}

	__device__ Reference& operator = (const Reference& other)
	{
		return *this = static_cast<T>(other);
	}
	template <typename U>
	__device__ Reference& operator = (const Reference<S, U>& other)
	{
		return *this = static_cast<U>(other);
	}

	__device__ Reference& operator += (const T& value)
	{
		return *this = *this + value;
	}
	__device__ Reference& operator |= (const T& value)
	{
		return *this = *this | value;
	}
	__device__ Reference& operator &= (const T& value)
	{
		return *this = *this & value;
	}
	__device__ Reference& operator *= (const T& value)
	{
		return *this = *this * value;
	}

	__device__ operator T() const
	{
		return load<S>(mPtr);
	}
	__device__ Reference& operator = (const T& value)
	{
		store<S>(mPtr, value);
		return *this;
	}

  //private:
	T* mPtr;

	__device__ explicit Reference(T& ref) : mPtr(&ref)
	{
	}

	template <typename U>
	friend __device__ void atomicAdd(Reference& ref, U value)
	{
		::atomicAdd(ref.mPtr, value);
	}
};

template <AddressSpace S, typename T>
struct Convert
{
	static __device__ T* from(T* ptr)
	{
		return ptr;
	}
	static __device__ T* to(T* ptr)
	{
		return ptr;
	}
};

#if CONVERT_ADDRESSES
template <typename T>
struct Convert<Shared, T>
{
	static __device__ T* from(T* ptr)
	{
		asm("cvta.shared." POINTER_TYPE " %0, %0;" : "+" POINTER_CONSTRAINT(ptr));
		return ptr;
	}
	static __device__ T* to(T* ptr)
	{
		asm("cvta.to.shared." POINTER_TYPE " %0, %0;" : "+" POINTER_CONSTRAINT(ptr));
		return ptr;
	}
};
template <typename T>
struct Convert<Global, T>
{
	static __device__ T* from(T* ptr)
	{
		asm("cvta.global." POINTER_TYPE " %0, %0;" : "+" POINTER_CONSTRAINT(ptr));
		return ptr;
	}
	static __device__ T* to(T* ptr)
	{
		asm("cvta.to.global." POINTER_TYPE " %0, %0;" : "+" POINTER_CONSTRAINT(ptr));
		return ptr;
	}
};
#endif

template <AddressSpace S, typename T>
__device__ T* generic(const Pointer<S, T>&);

// pointer forced to point to shared memory (only works for sizeof(T) <= 4)
template <AddressSpace S, typename T>
struct Pointer
{
	template <AddressSpace, typename> friend struct Pointer;
	friend __device__ T* generic<S, T>(const Pointer<S, T>&);
	friend struct GlobalParticleData;

	__device__ Pointer()
	{
	}
	__device__ Pointer(const Pointer& other) 
		: mPtr(other.mPtr)
	{
	}
	template <typename U>
	__device__ Pointer(const Pointer<S, U>& other)
	: mPtr(other.mPtr)
	{
	}

	// construct from generic pointer
	__device__ explicit Pointer(T* ptr) 
		: mPtr(Convert<S, T>::to(ptr))
	{
	}

	__device__ bool operator!=(const Pointer& other) const
	{
		return mPtr != other.mPtr;
	}
	__device__ bool operator<(const Pointer& other) const
	{
		return mPtr < other.mPtr;
	}

	__device__ Pointer operator + (ptrdiff_t i) const
	{
		return Pointer(*this) += i;
	}
	__device__ Pointer& operator += (ptrdiff_t i)
	{
		mPtr += i * stride();
		return *this;
	}
	__device__ Pointer operator - (ptrdiff_t i) const
	{
		return Pointer(*this) -= i;
	}
	__device__ Pointer& operator -= (ptrdiff_t i)
	{
		mPtr -= i * stride();
		return *this;
	}

	__device__ Pointer& operator ++ ()
	{
		mPtr += stride();
		return *this;
	}
	__device__ Pointer& operator -- ()
	{
		mPtr -= stride();
		return *this;
	}

	__device__ Reference<S, T> operator*() const
	{
		return Reference<S, T>(*mPtr);
	}
	__device__ Reference<S, T> operator[](int32_t i) const
	{
		return Reference<S, T>(mPtr[i * stride()]);
	}

  private:
	// convert back to generic pointer, private for safety, use generic() instead
	__device__ operator T*() const
	{
		return Convert<S, T>::from(mPtr);
	}

	__device__ static size_t stride() { return 1; }

	template <typename U>
	__device__ Pointer(const Pointer<S, U>& other, ptrdiff_t stridedOffset)
		: mPtr(other.mPtr + stridedOffset)
	{
	}

	T* mPtr;
};

// pointers to global memory are all referring to particle data
// stored as array of structs, so they have a stride of 4.
template<> __device__ size_t Pointer<Global, float>::stride() { return 4; }
template<> __device__ size_t Pointer<Global, const float>::stride() { return 4; }
template <AddressSpace S, typename T>
__device__ T* generic(const Pointer<S, T>& ptr)
{
	return ptr;
}

#if !CONVERT_ADDRESSES
template <typename T>
__device__ T* generic(T* ptr)
{
	return ptr;
}
#endif

/***************** Particle Data **********************/

template <typename T>
struct SharedParticleReference
{
	__device__ operator float3() const
	{
		float3 result;
		result.x = mReferences[0];
		result.y = mReferences[1];
		result.z = mReferences[2];
		return result;
	}

	__device__ SharedParticleReference& operator = (const float3& vec)
	{
		mReferences[0] = vec.x;
		mReferences[1] = vec.y;
		mReferences[2] = vec.z;
		return *this;
	}

	__device__ operator float4() const
	{
		float4 result;
		result.x = mReferences[0];
		result.y = mReferences[1];
		result.z = mReferences[2];
		result.w = mReferences[3];
		return result;
	}

	__device__ SharedParticleReference& operator = (const float4& vec)
	{
		mReferences[0] = vec.x;
		mReferences[1] = vec.y;
		mReferences[2] = vec.z;
		mReferences[3] = vec.w;
		return *this;
	}

	Reference<Shared, T> mReferences[4];
};

struct SharedParticleData
{
	typedef float3 VectorType;

	typedef Pointer<Shared, float> PointerType;
	typedef Pointer<Shared, const float> ConstPointerType;
	typedef Reference<Shared, float> ReferenceType;
	typedef Reference<Shared, const float> ConstReferenceType;

	typedef SharedParticleReference<float> ParticleReferenceType;
	typedef SharedParticleReference<const float> ParticleConstReferenceType;

	__device__ ReferenceType operator()(int32_t index, int32_t element)
	{
		return mPointers[element][index];
	}
	__device__ ConstReferenceType operator()(int32_t index, int32_t element) const
	{
		return mPointers[element][index];
	}

	__device__ ParticleReferenceType operator()(int32_t index)
	{
		ParticleReferenceType result = { mPointers[0][index], mPointers[1][index],
			                             mPointers[2][index], mPointers[3][index] };
		return result;
	}
	__device__ ParticleConstReferenceType operator()(int32_t index) const
	{
		ParticleConstReferenceType result = { mPointers[0][index], mPointers[1][index],
			                                  mPointers[2][index], mPointers[3][index] };
		return result;
	}

	__device__ const PointerType& operator[](int32_t element)
	{
		return mPointers[element];
	}
	__device__ ConstPointerType operator[](int32_t element) const
	{
		return mPointers[element];
	}

	PointerType mPointers[4];
};

template <typename T>
struct GlobalParticleReference
{
	__device__ GlobalParticleReference(Pointer<Global, T> ref) : mPtr(reinterpret_cast<T* const&>(ref))
	{
	}

#if CONVERT_ADDRESSES
	__device__ operator float4() const
	{
		float4 vec;
		asm("ld.global.v4.f32 {%0, %1, %2, %3}, [%4];"
		    : "=f"(vec.x), "=f"(vec.y), "=f"(vec.z), "=f"(vec.w)
		    : POINTER_CONSTRAINT(mPtr));
		return vec;
	}

	__device__ GlobalParticleReference& operator = (const float4& vec)
	{
		asm("st.global.v4.f32 [%0], {%1, %2, %3, %4};" ::POINTER_CONSTRAINT(mPtr), "f"(vec.x), "f"(vec.y), "f"(vec.z),
		    "f"(vec.w)
		    : "memory");
		return *this;
	}

	__device__ operator float3() const
	{
		float4 vec = *this;
		return make_float3(vec.x, vec.y, vec.z);
	}
#else

	__device__ operator float4() const
	{
		return *reinterpret_cast<const float4*>(mPtr);
	}

	__device__ GlobalParticleReference& operator = (const float4& vec)
	{
		*reinterpret_cast<float4*>(mPtr) = vec;
		return *this;
	}

	__device__ operator float3() const
	{
		return *reinterpret_cast<const float3*>(mPtr);
	}

	__device__ GlobalParticleReference& operator = (const float3& vec)
	{
		*reinterpret_cast<float3*>(mPtr) = vec;
		return *this;
	}
#endif

	T* mPtr; // pointer to global address
};

struct GlobalParticleData
{
#if CONVERT_ADDRESSES
	// ld.global.v4 saturates memory bandwidth better than 3x ld.global
	typedef float4 VectorType;
#else
	// the same isn't true for ld without state space
	typedef float3 VectorType;
#endif

	typedef Pointer<Global, float> PointerType;
	typedef Pointer<Global, const float> ConstPointerType;
	typedef Reference<Global, float> ReferenceType;
	typedef Reference<Global, const float> ConstReferenceType;

	typedef GlobalParticleReference<float> ParticleReferenceType;
	typedef GlobalParticleReference<const float> ParticleConstReferenceType;

	__device__ ReferenceType operator()(int32_t index, int32_t element)
	{
		return *PointerType(mPtr, index * 4 + element);
	}
	__device__ ConstReferenceType operator()(int32_t index, int32_t element) const
	{
		return *ConstPointerType(mPtr, index * 4 + element);
	}

	__device__ ParticleReferenceType operator()(int32_t index)
	{
		return PointerType(mPtr, index * 4);
	}

	__device__ ParticleConstReferenceType operator()(int32_t index) const
	{
		return ConstPointerType(mPtr, index * 4);
	}

	__device__ PointerType operator[](int32_t element)
	{
		return PointerType(mPtr, element);
	}
	__device__ ConstPointerType operator[](int32_t element) const
	{
		return ConstPointerType(mPtr, element);
	}

	PointerType mPtr;
};

/***************** Profiling **********************/
struct ProfileDisabledZone
{
	__device__ ProfileDisabledZone(cloth::CuProfileZoneIds::Enum)
	{
	}
};

#if defined(__CUDA_ARCH__) && defined(PX_PROFILE) // profile zones enabled for profile build

// code below is copied from GPUProfile.h and needs to be kept in sync.

#define NUM_WARPS_PER_PROFILE_BUFFER (4 * 1024 * 1024)

struct __align__(16) WarpProfileEvent
{
	__device__ WarpProfileEvent(uint16_t id)
	: block(blockIdx.x + gridDim.x * blockIdx.y), warp(threadIdx.x >> 5), userData(0), eventId(id)
	{
		uint32_t smid32, warpid32;
		asm volatile("mov.u32 %0, %smid;" : "=r"(smid32));
		asm volatile("mov.u32 %0, %warpid;" : "=r"(warpid32));
		asm volatile("mov.u32 %0, %clock;" : "=r"(startTime));
		smid = smid32;
		warpid = warpid32;
		endTime = startTime;
	}

	uint16_t block;
	uint8_t warp;
	uint8_t smid;
	uint8_t warpid;
	uint8_t userData;
	uint16_t eventId;
	uint32_t startTime;
	uint32_t endTime;
};

struct ProfileZone
{
	__device__ ProfileZone(cloth::CuProfileZoneIds::Enum id) : mEvent(0)
	{
		if (!gProfileBuffer || threadIdx.x & 0x1f)
			return;

		// +1: first entry reserved for counter
		uint32_t index = atomicAdd(reinterpret_cast<uint32_t*>(gProfileBuffer), 1) + 1;

		if (index >= NUM_WARPS_PER_PROFILE_BUFFER)
			return;

		mEvent = reinterpret_cast<WarpProfileEvent*>(gProfileBuffer) + index;

		new (mEvent) WarpProfileEvent(gProfileBaseId + id);
	}

	__device__ ~ProfileZone()
	{
		if (mEvent)
			mEvent->endTime = clock();
	}

	WarpProfileEvent* mEvent;
};

#else
typedef ProfileDisabledZone ProfileZone;
#endif

#if 1 // set to 1 to enable detailed profile zones
typedef ProfileZone ProfileDetailZone;
#else
typedef ProfileDisabledZone ProfileDetailZone;
#endif

namespace
{
// cut down version of thrust::uninitialized
// avoids warning about non-empty c'tor
template <typename T>
struct uninitialized
{
	__device__ inline T& get()
	{
		return *reinterpret_cast<T*>(data);
	}

	// maximum alignment required by device code is 16
	__align__(16) unsigned char data[sizeof(T)];
};
}

#if __CUDA_ARCH__ < 320
namespace
{
template <typename T>
__device__ T __ldg(const T* __restrict ptr)
{
	return *ptr;
}
}
#endif

#define CU_SOLVER_KERNEL_CU
#include "CuCollision.h"
#include "CuSelfCollision.h"

namespace
{
__device__ void loadIterData(const cloth::CuIterationData* __restrict iterData)
{
	if (threadIdx.x < gCuIterationDataSize)
	{
		gIterData.mIntegrationTrafo[threadIdx.x] = __ldg(iterData->mIntegrationTrafo + threadIdx.x);
	}
}

// integrate particle positions and store transposed
template <bool IsTurning, typename CurrentT, typename PreviousT>
__device__ void integrateParticles(CurrentT& current, PreviousT& previous)
{
	ProfileDetailZone zone(cloth::CuProfileZoneIds::INTEGRATE);

	const float* __restrict trafo = gIterData.mIntegrationTrafo;

	for (int32_t i = threadIdx.x; i < gClothData.mNumParticles; i += blockDim.x)
	{
		float4 prev = previous(i);
		float4 next = current(i);
		float4 cur = { next.x, next.y, next.z, prev.w };

		if (next.w == 0.0f)
			next.w = prev.w;

		if (next.w > 0.0f)
		{
			if (IsTurning)
			{
				next.x = next.x + trafo[3] + cur.x * trafo[15] + prev.x * trafo[6] + cur.y * trafo[16] +
				         prev.y * trafo[7] + cur.z * trafo[17] + prev.z * trafo[8];

				next.y = next.y + trafo[4] + cur.x * trafo[18] + prev.x * trafo[9] + cur.y * trafo[19] +
				         prev.y * trafo[10] + cur.z * trafo[20] + prev.z * trafo[11];

				next.z = next.z + trafo[5] + cur.x * trafo[21] + prev.x * trafo[12] + cur.y * trafo[22] +
				         prev.y * trafo[13] + cur.z * trafo[23] + prev.z * trafo[14];
			}
			else
			{
				next.x += (cur.x - prev.x) * trafo[6] + trafo[3];
				next.y += (cur.y - prev.y) * trafo[9] + trafo[4];
				next.z += (cur.z - prev.z) * trafo[12] + trafo[5];
			}

			cur.x += trafo[0];
			cur.y += trafo[1];
			cur.z += trafo[2];
		}

		current(i) = next;
		previous(i) = cur;
	}
}

template <typename CurrentT, typename PreviousT>
__device__ void integrateParticles(CurrentT& current, PreviousT& previous)
{
	if (gIterData.mIsTurning)
		integrateParticles<true>(current, previous);
	else
		integrateParticles<false>(current, previous);
}

template <typename CurrentT>
__device__ void accelerateParticles(CurrentT& current)
{
	// might be better to move this into integrate particles
	const float* __restrict accelerations = gFrameData.mParticleAccelerations;

	if (!accelerations)
		return;

	ProfileDetailZone zone(cloth::CuProfileZoneIds::ACCELERATE);

	__syncthreads(); // looping with 4 instead of 1 thread per particle

	float sqrIterDt = ~threadIdx.x & 0x3 ? gFrameData.mIterDt * gFrameData.mIterDt : 0.0f;
	typename CurrentT::PointerType sharedCurPos = current[threadIdx.x % 4];

	for (int32_t i = threadIdx.x; i < gClothData.mNumParticles * 4; i += blockDim.x)
	{
		// turning this into __ldg slows kernel down even without particle accelerations (!)
		if (current(i / 4, 3) > 0.0f)
			sharedCurPos[i / 4] += accelerations[i] * sqrIterDt;
	}

	__syncthreads();
}

__device__ float3 operator + (const float3& u, const float3& v)
{
	return make_float3(u.x + v.x, u.y + v.y, u.z + v.z);
}
__device__ float3 operator - (const float3& u, const float3& v)
{
	return make_float3(u.x - v.x, u.y - v.y, u.z - v.z);
}
__device__ float3 operator*(float s, const float3& v)
{
	return make_float3(v.x * s, v.y * s, v.z * s);
}
__device__ float dot3(const float3& u, const float3& v)
{
	return u.x * v.x + u.y * v.y + u.z * v.z;
}
__device__ float3 cross3(const float3& u, const float3& v)
{
	return make_float3(u.y * v.z - u.z * v.y, u.z * v.x - u.x * v.z, u.x * v.y - u.y * v.x);
}
__device__ void applyImpulse(SharedParticleData::ParticleReferenceType pos, const float3& impulse)
{
	float scale = -pos.mReferences[3];

#if CONVERT_ADDRESSES
	// Use this instead of atomicAdd function to work around compiler issue treating the pointer as global memory instead of shared memory
	asm("red.shared.add.f32 [%0], %1;" ::POINTER_CONSTRAINT(pos.mReferences[0].mPtr), "f"(impulse.x * scale));
	asm("red.shared.add.f32 [%0], %1;" ::POINTER_CONSTRAINT(pos.mReferences[1].mPtr), "f"(impulse.y * scale));
	asm("red.shared.add.f32 [%0], %1;" ::POINTER_CONSTRAINT(pos.mReferences[2].mPtr), "f"(impulse.z * scale));
#else
	atomicAdd(pos.mReferences[0].mPtr, impulse.x * scale);
	atomicAdd(pos.mReferences[1].mPtr, impulse.y * scale);
	atomicAdd(pos.mReferences[2].mPtr, impulse.z * scale);
#endif	
}
__device__ void applyImpulse(GlobalParticleData::ParticleReferenceType pos, const float3& impulse)
{
	float scale = -pos.mPtr[3];
	atomicAdd(pos.mPtr + 0, impulse.x * scale);
	atomicAdd(pos.mPtr + 1, impulse.y * scale);
	atomicAdd(pos.mPtr + 2, impulse.z * scale);
}

template <bool IsTurning, typename CurrentT, typename PreviousT>
__device__ void applyWind(CurrentT& current, PreviousT& previous)
{
	const float dragCoefficient = gFrameData.mDragCoefficient;
	const float liftCoefficient = gFrameData.mLiftCoefficient;
	const float fluidDensity = gFrameData.mFluidDensity;
	const float itrDt = gFrameData.mIterDt;

	if (dragCoefficient == 0.0f && liftCoefficient == 0.0f)
		return;

	ProfileDetailZone zone(cloth::CuProfileZoneIds::WIND);

	const float oneThird = 1 / 3.0f;
	float3 wind = make_float3(gIterData.mWind[0], gIterData.mWind[1], gIterData.mWind[2]);

	const uint16_t* tIt = gClothData.mTriangles;
	for (int32_t i = threadIdx.x; i < gClothData.mNumTriangles; i += blockDim.x)
	{
		uint16_t i0 = tIt[i * 3 + 0];
		uint16_t i1 = tIt[i * 3 + 1];
		uint16_t i2 = tIt[i * 3 + 2];

		float3 c0 = current(i0);
		float3 c1 = current(i1);
		float3 c2 = current(i2);

	//	float w1 = current(i0, 3);
	//	float w2 = current(i1, 3);
	//	float w2 = current(i2, 3);
	//
	//	float wMult = w1 * w2 * w3;
	//	float invMass = wMult < FLT_EPSILON ? 0.f : w1 * w2 * w3 / (w1 * w2 + w1 * w3 + w2 * w3);

		float3 p0 = previous(i0);
		float3 p1 = previous(i1);
		float3 p2 = previous(i2);

		float3 cur = oneThird * (c0 + c1 + c2);
		float3 prev = oneThird * (p0 + p1 + p2);

		float3 delta = cur - prev + wind;

		if (IsTurning)
		{
			const float3* rot = reinterpret_cast<const float3*>(gFrameData.mRotation);
			float3 d = wind - prev;
			delta = cur + d.x * rot[0] + d.y * rot[1] + d.z * rot[2];
		}

		float3 normal = cross3(c2 - c0, c1 - c0);

		const float doubleArea = sqrtf(dot3(normal, normal));
		normal = (1.0f / doubleArea) * normal;

		float invSqrScale = dot3(delta, delta);
		float scale = rsqrtf(invSqrScale);
		float deltaLength = sqrtf(invSqrScale);

		float cosTheta = dot3(normal, delta) * scale;
		float sinTheta = sqrtf(max(0.0f, 1.0f - cosTheta * cosTheta));

		float3 liftDir = cross3(cross3(delta, normal), scale * delta);

		float3 lift = liftCoefficient * cosTheta * sinTheta * ((deltaLength / itrDt) * liftDir);
		float3 drag = dragCoefficient * abs(cosTheta) * ((deltaLength / itrDt) * delta);

		float3 impulse = invSqrScale < FLT_EPSILON ? make_float3(0.0f, 0.0f, 0.0f) : fluidDensity * doubleArea * (lift + drag);

		applyImpulse(current(i0), impulse);
		applyImpulse(current(i1), impulse);
		applyImpulse(current(i2), impulse);
	}

	__syncthreads();
}

template <typename CurrentT, typename PreviousT>
__device__ void applyWind(CurrentT& current, PreviousT& previous)
{
	if (gIterData.mIsTurning)
		applyWind<true>(current, previous);
	else
		applyWind<false>(current, previous);
}

template <typename CurrentT>
__device__ void constrainTether(CurrentT& current)
{
	if (0.0f == gFrameData.mTetherConstraintStiffness || !gClothData.mNumTethers)
		return;

	ProfileDetailZone zone(cloth::CuProfileZoneIds::TETHER);

	int32_t numParticles = gClothData.mNumParticles;
	int32_t numTethers = gClothData.mNumTethers;
	assert(0 == numTethers % numParticles);

	float stiffness = numParticles * __fdividef(gFrameData.mTetherConstraintStiffness, numTethers);
	float scale = gClothData.mTetherConstraintScale;

	const uint32_t* __restrict tIt = reinterpret_cast<const uint32_t*>(gClothData.mTethers);

	for (int32_t i = threadIdx.x; i < numParticles; i += blockDim.x)
	{
		float posX = current(i, 0);
		float posY = current(i, 1);
		float posZ = current(i, 2);

		float offsetX = 0.0f;
		float offsetY = 0.0f;
		float offsetZ = 0.0f;

		for (int32_t j = i; j < numTethers; j += gClothData.mNumParticles)
		{
			uint32_t tether = __ldg(tIt + j);

			int32_t anchor = tether & 0xffff;
			float deltaX = current(anchor, 0) - posX;
			float deltaY = current(anchor, 1) - posY;
			float deltaZ = current(anchor, 2) - posZ;

			float sqrLength = FLT_EPSILON + deltaX * deltaX + deltaY * deltaY + deltaZ * deltaZ;

			float radius = (tether >> 16) * scale;
			float slack = 1.0f - radius * rsqrtf(sqrLength);

			if (slack > 0.0f)
			{
				offsetX += deltaX * slack;
				offsetY += deltaY * slack;
				offsetZ += deltaZ * slack;
			}
		}

		current(i, 0) = posX + offsetX * stiffness;
		current(i, 1) = posY + offsetY * stiffness;
		current(i, 2) = posZ + offsetZ * stiffness;
	}
}

template <typename CurrentT>
__device__ void solveFabric(CurrentT& current)
{
	ProfileDetailZone zone(cloth::CuProfileZoneIds::FABRIC);

	const cloth::CuPhaseConfig* __restrict cIt = (cloth::CuPhaseConfig*)gSharedMemory;
	const cloth::CuPhaseConfig* cEnd = cIt + gClothData.mNumPhases;

	for (; cIt != cEnd; ++cIt)
	{
		__syncthreads();

		ProfileDetailZone zone(cloth::CuProfileZoneIds::CONSTRAINT_SET);

		int32_t numConstraints = cIt->mNumConstraints;
		if (threadIdx.x >= numConstraints)
			continue;

		const uint32_t* __restrict iIt = reinterpret_cast<const uint32_t*>(cIt->mIndices) + threadIdx.x;
		const float* restvalues = cIt->mRestvalues;
		const float* rIt = restvalues + threadIdx.x;
		const float* rEnd = restvalues + numConstraints;

		const float* stIt = cIt->mStiffnessValues + threadIdx.x;
		bool useStiffnessPerConstraint = cIt->mStiffnessValues!=nullptr;

		uint32_t vpijPrefetch = __ldg(iIt);
		float rijPrefetch = __ldg(rIt);
		float stijPrefetch;
		if (useStiffnessPerConstraint)
			stijPrefetch = __ldg(stIt);

		float stiffness = cIt->mStiffness;
		float stiffnessMultiplier = cIt->mStiffnessMultiplier;
		float compressionLimit = cIt->mCompressionLimit;
		float stretchLimit = cIt->mStretchLimit;

		do
		{
			rIt += blockDim.x;
			iIt += blockDim.x;
			stIt += blockDim.x;

			int32_t vpi = USHRT_MAX & vpijPrefetch;
			int32_t vpj = USHRT_MAX & vpijPrefetch >> 16;
			float rij = rijPrefetch;
			float stij = useStiffnessPerConstraint?1.0f - exp2f(stijPrefetch * gFrameData.mStiffnessExponent):stiffness;

			if (rIt < rEnd)
			{
				vpijPrefetch = __ldg(iIt);
				rijPrefetch = __ldg(rIt);
				if (useStiffnessPerConstraint)
					stijPrefetch = __ldg(stIt);
			}

			float vxi = current(vpi, 0);
			float vyi = current(vpi, 1);
			float vzi = current(vpi, 2);
			float vwi = current(vpi, 3);

			float vxj = current(vpj, 0);
			float vyj = current(vpj, 1);
			float vzj = current(vpj, 2);
			float vwj = current(vpj, 3);

			float hxij = vxj - vxi;
			float hyij = vyj - vyi;
			float hzij = vzj - vzi;

			float e2ij = FLT_EPSILON + hxij * hxij + hyij * hyij + hzij * hzij;
			float negErij = rij > FLT_EPSILON ? -1.0f + rij * rsqrtf(e2ij) : 0.0f;

			negErij = negErij + stiffnessMultiplier * max(compressionLimit, min(-negErij, stretchLimit));

			float negExij = __fdividef(negErij * stij, FLT_EPSILON + vwi + vwj);

			float vmi = -vwi * negExij;
			current(vpi, 0) = vxi + vmi * hxij;
			current(vpi, 1) = vyi + vmi * hyij;
			current(vpi, 2) = vzi + vmi * hzij;

			float vmj = +vwj * negExij;
			current(vpj, 0) = vxj + vmj * hxij;
			current(vpj, 1) = vyj + vmj * hyij;
			current(vpj, 2) = vzj + vmj * hzij;

		} while (rIt < rEnd);
	}

	__syncthreads();
}

template <typename CurrentT>
__device__ void constrainMotion(CurrentT& current, float alpha)
{
	if (!gFrameData.mStartMotionConstraints)
		return;

	ProfileDetailZone zone(cloth::CuProfileZoneIds::MOTION);

	// negative because of fused multiply-add optimization
	float negativeScale = -gClothData.mMotionConstraintScale;
	float negativeBias = -gClothData.mMotionConstraintBias;

	const float4* startIt = reinterpret_cast<const float4*>(gFrameData.mStartMotionConstraints);
	const float4* targetIt = reinterpret_cast<const float4*>(gFrameData.mTargetMotionConstraints);

	for (int32_t i = threadIdx.x; i < gClothData.mNumParticles; i += blockDim.x)
	{
		float4 startPos = __ldg(startIt + i);
		float4 targetPos = __ldg(targetIt + i);

		float sphereX = startPos.x + (targetPos.x - startPos.x) * alpha;
		float sphereY = startPos.y + (targetPos.y - startPos.y) * alpha;
		float sphereZ = startPos.z + (targetPos.z - startPos.z) * alpha;
		float sphereW = startPos.w + (targetPos.w - startPos.w) * alpha;

		float dx = sphereX - current(i, 0);
		float dy = sphereY - current(i, 1);
		float dz = sphereZ - current(i, 2);

		float sqrLength = FLT_EPSILON + dx * dx + dy * dy + dz * dz;
		float negativeRadius = min(0.0f, sphereW * negativeScale + negativeBias);

		float slack = max(negativeRadius * rsqrtf(sqrLength) + 1.0f, 0.0f) * gFrameData.mMotionConstraintStiffness;

		current(i, 0) += slack * dx;
		current(i, 1) += slack * dy;
		current(i, 2) += slack * dz;

		// set invMass to zero if radius is zero
		if (negativeRadius >= 0.0f)
			current(i, 3) = 0.0f;
	}
}

template <typename T>
__device__ void constrainSeparation(T& current, float alpha)
{
	if (!gFrameData.mStartSeparationConstraints)
		return;

	ProfileDetailZone zone(cloth::CuProfileZoneIds::SEPARATION);

	const float4* startIt = reinterpret_cast<const float4*>(gFrameData.mStartSeparationConstraints);
	const float4* targetIt = reinterpret_cast<const float4*>(gFrameData.mTargetSeparationConstraints);

	for (int32_t i = threadIdx.x; i < gClothData.mNumParticles; i += blockDim.x)
	{
		float4 startPos = __ldg(startIt + i);
		float4 targetPos = __ldg(targetIt + i);

		float sphereX = startPos.x + (targetPos.x - startPos.x) * alpha;
		float sphereY = startPos.y + (targetPos.y - startPos.y) * alpha;
		float sphereZ = startPos.z + (targetPos.z - startPos.z) * alpha;
		float sphereW = startPos.w + (targetPos.w - startPos.w) * alpha;

		float dx = sphereX - current(i, 0);
		float dy = sphereY - current(i, 1);
		float dz = sphereZ - current(i, 2);

		float sqrLength = FLT_EPSILON + dx * dx + dy * dy + dz * dz;

		float slack = min(0.0f, 1.0f - sphereW * rsqrtf(sqrLength));

		current(i, 0) += slack * dx;
		current(i, 1) += slack * dy;
		current(i, 2) += slack * dz;
	}
}

template <typename CurrentT, typename PreviousT>
__device__ void updateSleepState(const CurrentT& current, const PreviousT& previous)
{
	ProfileDetailZone zone(cloth::CuProfileZoneIds::SLEEP);

	if (!threadIdx.x)
		gFrameData.mSleepTestCounter += max(1, uint32_t(gFrameData.mIterDt * 1000));

	__syncthreads();

	if (gFrameData.mSleepTestCounter < gClothData.mSleepTestInterval)
		return;

	float maxDelta = 0.0f;
	for (int32_t i = threadIdx.x; i < gClothData.mNumParticles; i += blockDim.x)
	{
		float4 prev = previous(i);
		maxDelta = max(fabsf(current(i, 0) - prev.x), maxDelta);
		maxDelta = max(fabsf(current(i, 1) - prev.y), maxDelta);
		maxDelta = max(fabsf(current(i, 2) - prev.z), maxDelta);
	}

	if (!threadIdx.x)
	{
		++gFrameData.mSleepPassCounter;
		gFrameData.mSleepTestCounter -= gClothData.mSleepTestInterval;
	}

	__syncthreads();

	if (maxDelta > gClothData.mSleepThreshold * gFrameData.mIterDt)
		gFrameData.mSleepPassCounter = 0;
}

template <typename CurrentT, typename PreviousT>
__device__ void simulateCloth(CurrentT& current, PreviousT& previous)
{
	// apply exponent to phase configs
	assert(blockDim.x >= gClothData.mNumPhases);
	if (threadIdx.x < gClothData.mNumPhases)
	{
		float exponent = gFrameData.mStiffnessExponent;
		float* ptr = gSharedMemory + threadIdx.x * gCuPhaseConfigSize;
		ptr[0] = 1.0f - exp2f(ptr[0] * exponent);
		ptr[1] = 1.0f - exp2f(ptr[1] * exponent);
	}

	uint32_t numIterations = gFrameData.mNumIterations;
	float invNumIterations = __fdividef(1.0f, numIterations);

	const cloth::CuIterationData* iterData = gFrameData.mIterationData;
	const cloth::CuIterationData* iterEnd = iterData + numIterations;

	loadIterData(iterData);

	__syncthreads();

	for (float alpha = invNumIterations; iterData++ != iterEnd; alpha += invNumIterations)
	{
		integrateParticles(current, previous);
		accelerateParticles(current);
		applyWind(current, previous);
		constrainMotion(current, alpha);
		constrainTether(current);
		solveFabric(current);
		loadIterData(iterData);
		constrainSeparation(current, alpha);
		gCollideParticles.get()(current, previous, alpha);
		gSelfCollideParticles.get()(current);
		updateSleepState(current, previous);
	}

	__syncthreads();
}

__device__ void simulateShared()
{
	ProfileZone zone(cloth::CuProfileZoneIds::SIMULATE_SHARED);

	__shared__ uninitialized<SharedParticleData> current;
	__shared__ uninitialized<SharedParticleData> previous;

	int32_t configDataSize = gClothData.mNumPhases * gCuPhaseConfigSize;
	int32_t particlesDataSize = 4 * gClothData.mNumParticles;

	Pointer<Shared, float> sharedCurPos =
	    Pointer<Shared, float>(gSharedMemory + configDataSize + threadIdx.x % 4 * gClothData.mNumParticles);
	Pointer<Shared, float> sharedPrevPos = sharedCurPos + particlesDataSize;

	if (threadIdx.x < 4)
	{
		current.get().mPointers[threadIdx.x] = sharedCurPos;
		previous.get().mPointers[threadIdx.x] = sharedPrevPos;
	}

	float* globalCurPos = gClothData.mParticles;
	float* globalPrevPos = gClothData.mParticles + particlesDataSize;

	// copy particles from device memory to shared memory and transpose
	for (int32_t i = threadIdx.x; i < particlesDataSize; i += blockDim.x)
	{
		sharedCurPos[i / 4] = globalCurPos[i];
		sharedPrevPos[i / 4] = globalPrevPos[i];
	}

	simulateCloth(current.get(), previous.get());

	// copy particles from shared memory to device memory and transpose
	for (int32_t i = threadIdx.x; i < particlesDataSize; i += blockDim.x)
	{
		globalCurPos[i] = sharedCurPos[i / 4];
		globalPrevPos[i] = sharedPrevPos[i / 4];
	}

	__syncthreads();
}

__device__ void simulateStreamed()
{
	ProfileZone zone(cloth::CuProfileZoneIds::SIMULATE_STREAMED);

	__shared__ uninitialized<SharedParticleData> current;
	__shared__ uninitialized<GlobalParticleData> previous;

	int32_t configDataSize = gClothData.mNumPhases * gCuPhaseConfigSize;
	int32_t particlesDataSize = 4 * gClothData.mNumParticles;

	float* globalCurPos = gClothData.mParticles;
	Pointer<Shared, float> sharedCurPos =
	    Pointer<Shared, float>(gSharedMemory + configDataSize + threadIdx.x % 4 * gClothData.mNumParticles);

	if (threadIdx.x < 4)
		current.get().mPointers[threadIdx.x] = sharedCurPos;
	if (!threadIdx.x)
		previous.get().mPtr = GlobalParticleData::PointerType(globalCurPos + particlesDataSize);

	// copy particles from device memory to shared memory and transpose
	for (int32_t i = threadIdx.x; i < particlesDataSize; i += blockDim.x)
		sharedCurPos[i / 4] = globalCurPos[i];

	simulateCloth(current.get(), previous.get());

	// copy particles from shared memory to device memory and transpose
	for (int32_t i = threadIdx.x; i < particlesDataSize; i += blockDim.x)
		globalCurPos[i] = sharedCurPos[i / 4];

	__syncthreads();
}

__device__ void simulateGlobal()
{
	ProfileZone zone(cloth::CuProfileZoneIds::SIMULATE_GLOBAL);

	__shared__ uninitialized<GlobalParticleData> current;
	__shared__ uninitialized<GlobalParticleData> previous;

	if (!threadIdx.x)
	{
		GlobalParticleData::PointerType globalCurPos(gClothData.mParticles);
		current.get().mPtr = globalCurPos;
		previous.get().mPtr = globalCurPos + gClothData.mNumParticles;
	}

	simulateCloth(current.get(), previous.get());
}

} // anonymous namespace

extern "C" __global__ void
#if __CUDA_ARCH__ >= 300
__launch_bounds__(1024, 1)
#else
__launch_bounds__(512, 1)
#endif
    simulateCloths(cloth::CuKernelData kernelData)
{
	gProfileBuffer = kernelData.mProfileBuffer;
	gProfileBaseId = kernelData.mProfileBaseId;

	ProfileZone zone(cloth::CuProfileZoneIds::SIMULATE);

	// check that http://nvbugs/1038473 is fixed
	assert(gSharedMemory > (float*)&gFrameData);
	assert(gSharedMemory > (float*)&gClothData);

	// fetch cloth index from queue
	__shared__ uint32_t clothIdx;
	if (!threadIdx.x)
		clothIdx = atomicInc(kernelData.mClothIndex, gridDim.x - 1);
	__syncthreads();
	assert(clothIdx < gridDim.x);

	// copy cloth data to shared memory
	const uint32_t* clothData = reinterpret_cast<const uint32_t*>(kernelData.mClothData + clothIdx);
	if (threadIdx.x < gCuClothDataSize)
		reinterpret_cast<uint32_t*>(&gClothData)[threadIdx.x] = clothData[threadIdx.x];

	// copy frame data to shared memory
	uint32_t* frameData = reinterpret_cast<uint32_t*>(kernelData.mFrameData + clothIdx);
	if (threadIdx.x < gCuFrameDataSize)
		reinterpret_cast<uint32_t*>(&gFrameData)[threadIdx.x] = frameData[threadIdx.x];

	__syncthreads();

	if (gFrameData.mSleepPassCounter >= gClothData.mSleepAfterCount)
		return; // cloth is sleeping, exit

	// copy phase configs to shared memory
	int32_t configDataSize = gClothData.mNumPhases * gCuPhaseConfigSize;
	for (int32_t i = threadIdx.x; i < configDataSize; i += blockDim.x)
		gSharedUnsigned[i] = reinterpret_cast<const uint32_t*>(gClothData.mPhaseConfigs)[i];


	Pointer<Shared, uint32_t> scratchPtr = Pointer<Shared, uint32_t>(
	    gSharedUnsigned + configDataSize + 4 * gFrameData.mNumSharedPositions * gClothData.mNumParticles);

	// initialize with placement new
	new (gCollideParticles.data) CuCollision(scratchPtr);
	new (gSelfCollideParticles.data) CuSelfCollision();

	// copy particles and constraints to device
	if (gFrameData.mDeviceParticlesDirty)
	{
		for (int32_t i = threadIdx.x; i < gClothData.mNumParticles * 8; i += blockDim.x)
			gClothData.mParticles[i] = gClothData.mParticlesHostCopy[i];
	}
	if (gFrameData.mHostMotionConstraints)
	{
		for (int32_t i = threadIdx.x; i < gClothData.mNumParticles * 4; i += blockDim.x)
			gFrameData.mTargetMotionConstraints[i] = gFrameData.mHostMotionConstraints[i];
	}
	if (gFrameData.mHostSeparationConstraints)
	{
		for (int32_t i = threadIdx.x; i < gClothData.mNumParticles * 4; i += blockDim.x)
			gFrameData.mTargetSeparationConstraints[i] = gFrameData.mHostSeparationConstraints[i];
	}
	if (gFrameData.mHostParticleAccelerations)
	{
		for (int32_t i = threadIdx.x; i < gClothData.mNumParticles * 4; i += blockDim.x)
			gFrameData.mParticleAccelerations[i] = gFrameData.mHostParticleAccelerations[i];
	}

	// necessary to ensure phase configs are fully loaded before setup in simulateCloth()
	__syncthreads();

	switch(gFrameData.mNumSharedPositions)
	{
	case 0:
		simulateGlobal();
		break;
	case 1:
		simulateStreamed();
		break;
	case 2:
		simulateShared();
		break;
	}

	// write back frame data
	if (threadIdx.x < gCuFrameDataSize)
		frameData[threadIdx.x] = reinterpret_cast<const uint32_t*>(&gFrameData)[threadIdx.x];

	// copy particles to host
	for (int32_t i = threadIdx.x; i < gClothData.mNumParticles * 8; i += blockDim.x)
		gClothData.mParticlesHostCopy[i] = gClothData.mParticles[i];
}

const char* cloth::getKernelFunctionName()
{
	return "simulateCloths";
}
